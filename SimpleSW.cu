#include "hip/hip_runtime.h"
#include<hip/hip_runtime.h>
#include<nvvm.h>
#include<unistd.h>
#include<malloc.h>

#include<iostream>
#include<iomanip>

#include"SimpleSW.h"
#include"Data.h"
#include"Cost.h"


// Lenght of each data
__constant__ int gcT_size;
__constant__ int gcP_size;

// Threshold of the SW algorithm
__constant__ int gcThre;

// Data of the query
__constant__ char gcP_seq[1024];

// Cost and Gain
__constant__ int gcMatch;
__constant__ int gcMiss;
__constant__ int gcExtend;
__constant__ int gcBegin;

enum{
	Zero,
	Diagonal,
	Vertical,
	Horizon,
};

using namespace std;

SimpleSW::SimpleSW(const Data& db, const Data& q, int threshold, int block_num) : mBlock{ block_num }
{
	cout << "Simple SW algorithm start" << endl;
	// Sieze check
	if(q.size() > 1024 || db.size() > 1024 * 1024 * 1024){
		cout << "Too large size" << endl;
		return;
	}
	mThreshold = threshold;
	// Set value in constant memory
	mSize = db.size();
	int psize = q.size();
	hipMemcpyToSymbol(HIP_SYMBOL(gcT_size), &mSize, sizeof(int));
	hipMemcpyToSymbol(HIP_SYMBOL(gcP_size), &psize, sizeof(int));
	hipMemcpyToSymbol(HIP_SYMBOL(gcThre), &threshold, sizeof(int));
	hipMemcpyToSymbol(HIP_SYMBOL(gcP_seq), q.data(), sizeof(char) * q.size());
	// Cost and gain
	int gain = cost.Match;
	hipMemcpyToSymbol(HIP_SYMBOL(gcMatch), &gain, sizeof(int));
	gain = cost.Miss;
	hipMemcpyToSymbol(HIP_SYMBOL(gcMiss), &gain, sizeof(int));
	gain = cost.Extend;
	hipMemcpyToSymbol(HIP_SYMBOL(gcExtend), &gain, sizeof(int));
	gain = cost.Begin;
	hipMemcpyToSymbol(HIP_SYMBOL(gcBegin), &gain, sizeof(int));
	// Dynamic Programing part by call_DP
	call_DP(db, q);
	cout << "Max score is " << max_score() << ", max position is " << max_position() << endl; 
	cout << "Simple SW algorithm end" << endl;
}

SimpleSW::~SimpleSW(){
	if(mScore){
		delete[] mScore;
		mScore = nullptr;
	}
}

// Implementation 
// No traceback
__global__ void DP(char* dT_seq, int* dScore, int block_size){
	// ThreadId = q point
	int id = threadIdx.x;
	// The acid in this thread
	char p = gcP_seq[id];
	// p-1 row line's value
	__shared__ int Hp_1[1024];
	__shared__ int Ep_1[1024];	
	// Temporary
	int Hp_1_buf = 0;
	int Ep_1_buf = 0;
	// t-1 element value
	int Ht_1 = 0;
	int Ft_1 = 0;
	// p-1 t-1 element value
	int Ht_1p_1 = 0;
	// Initialize
	Hp_1[id] = 0;
	Ep_1[id] = 0;
	// Similar score
	int sim = 0;
	// Set start point
	int start = (block_size - gcP_size) * blockIdx.x;
	if(blockIdx.x != 0){ start -= gcP_size; }	// Take margin
	// Culcurate elements
	for(int t = start - id; t < start + block_size; ++t){
		// Control culcurate order
		if(t<0 || t < start){}
		// Get similar score
		else{
			// Compare acids
			if(dT_seq[t] == p){sim = gcMatch;}
			else{sim = gcMiss;}
		}
		// SW algorithm
		// Culcurate each elements
		Ht_1p_1 += sim;	// Diagonal
		Ht_1 += gcBegin;	// Horizon (Start)
		Ft_1 += gcExtend;	// Horizon (Extend)
		Hp_1_buf = Hp_1[id] + gcBegin;	// Vertical (Start)
		Ep_1_buf = Ep_1[id] + gcExtend;	// Vertical (Extend)
		// Choose the gap score
		if(Ht_1 > Ft_1){Ft_1 = Ht_1;}	// Horizon
		if(Hp_1_buf > Ft_1){Ep_1_buf = Hp_1_buf;}	// Vertical
		// Choose the max score
		// Ht_1 is stored the max score
		if(Ht_1p_1 > Ep_1_buf){
			// Diagonal
			if(Ht_1p_1 > Ft_1){
				Ht_1 = Ht_1p_1;
			}
			// Horizon
			else{
				Ht_1 = Ft_1;
			}
		}
		else {
			// Vertical
			if(Ep_1_buf > Ft_1){
				Ht_1 = Ep_1_buf;
			}
			// Horizon
			else{
				Ht_1 = Ft_1;
			}
		}
		// The case 0 is max
		if(Ht_1 <= 0){
			Ht_1 = 0;
			// Set 0 other value
			Ft_1 = 0;
			Ep_1_buf = 0;
		}
		// Hp-1 is next Ht-1p-1 
		Ht_1p_1 = Hp_1[id];
		__syncthreads();
		// Set value need next culcurate
		// p+1 row line
		if(t >= 0){
			Hp_1[id + 1] = Ht_1;
			Ep_1[id + 1] = Ep_1_buf;
		}
		if(Ht_1 >= gcThre){
			if(Ht_1 >= dScore[t]){
			// Set score
				dScore[t] = Ht_1;
			}
		} 
		__syncthreads();
		// for end
	}
}

// With traceback
__global__ void DPwith(char* dT_seq, char* dTrace, int start, int length){
	// ThreadId = q point
	int id = threadIdx.x;
	// The acid in this thread
	char p = gcP_seq[id];
	// p-1 row line's value
	__shared__ int Hp_1[1024];
	__shared__ int Ep_1[1024];	
	// Temporary
	int Hp_1_buf = 0;
	int Ep_1_buf = 0;
	// t-1 element value
	int Ht_1 = 0;
	int Ft_1 = 0;
	// p-1 t-1 element value
	int Ht_1p_1 = 0;
	// Initialize
	Hp_1[id] = 0;
	Ep_1[id] = 0;
	// Similar score
	int sim = 0;
	int point = id * length - id;
	// Culcurate elements
	for(int t = -id + start; t < start + length; ++t){
		// Control culcurate order
		if(t<start){}
		// Get similar score
		else{
			// Compare acids
			if(dT_seq[t] == p){sim = gcMatch;}
			else{sim = gcMiss;}
		}
		// SW algorithm
		// Culcurate each elements
		Ht_1p_1 += sim;	// Diagonal
		Ht_1 += gcBegin;	// Horizon (Start)
		Ft_1 += gcExtend;	// Horizon (Extend)
		Hp_1_buf = Hp_1[id] + gcBegin;	// Vertical (Start)
		Ep_1_buf = Ep_1[id] + gcExtend;	// Vertical (Extend)
		// Choose the gap score
		if(Ht_1 > Ft_1){Ft_1 = Ht_1;}	// Horizon
		if(Hp_1_buf > Ft_1){Ep_1_buf = Hp_1_buf;}	// Vertical
		// Choose the max score
		// Ht_1 is stored the max score
		if(Ht_1p_1 > Ep_1_buf){
			// Diagonal
			if(Ht_1p_1 > Ft_1){
				Ht_1 = Ht_1p_1;
				dTrace[point] = Diagonal;
			}
			// Horizon
			else{
				Ht_1 = Ft_1;
				dTrace[point] = Horizon;
			}
		}
		else {
			// Vertical
			if(Ep_1_buf > Ft_1){
				Ht_1 = Ep_1_buf;
				dTrace[point] = Vertical;
			}
			// Horizon
			else{
				Ht_1 = Ft_1;
				dTrace[point] = Horizon;
			}
		}
		// The case 0 is max
		if(Ht_1 <= 0){
			Ht_1 = 0;
			// Set 0 other value
			Ft_1 = 0;
			Ep_1_buf = 0;
			dTrace[point] = Zero;
		}
		// Hp-1 is next Ht-1p-1 
		Ht_1p_1 = Hp_1[id];
		__syncthreads();
		// Set value need next culcurate
		// p+1 row line
		if(t >= start){
			Hp_1[id + 1] = Ht_1;
			Ep_1[id + 1] = Ep_1_buf;
			// DEBUG, score check
	//		dTrace[point] = (char)(Ht_1);
		} 
		++point;
		__syncthreads();
		// for end
	}
}

// Provisional
void SimpleSW::call_DP(const Data& db, const Data& q){
	// Set db
	char* dT_seq;
	hipMalloc((void**)&dT_seq, sizeof(char)*db.size());
	hipMemcpy(dT_seq, db.data(), sizeof(char)*db.size(), hipMemcpyHostToDevice);
	// Set block size
	int block_size = (db.size()/mBlock) + q.size();
	// Set Score and point
	int* dScore;
	hipMalloc((void**)&dScore, sizeof(int)*db.size());
	int* init0 = new int[db.size()];
	for(int i=0;i<db.size();++i){init0[i]=0;}
	hipMemcpy(dScore, init0, sizeof(int)*db.size(), hipMemcpyHostToDevice);
	// Main process
	DP<<<mBlock,q.size()>>>(dT_seq, dScore, block_size);	
	// Score and point copy
	mScore = new int[db.size()];
	hipMemcpy(mScore, dScore, sizeof(int)*db.size(), hipMemcpyDeviceToHost);
	// traceback if db has homelogy
//	checkScore(db, q);
	delete[] init0;
	hipFree(dT_seq);
	hipFree(dScore);
}

// score -> 0~16 : 17~31 = score : point of q
void SimpleSW::checkScore(const Data& db, const Data& q){
	// get the max score
	int max = max_score();
	int x = max_position();
	if(max != 0){
		// Call DP in limit range
		// Set db
		char* dT_seq;
		hipMalloc((void**)&dT_seq, sizeof(char)*db.size());	// TODO Too much range
		hipMemcpy(dT_seq, db.data(), sizeof(char)*db.size(), hipMemcpyHostToDevice);
		// Set Traceback
		int length = q.size() * 2;	// Enough
		char* dTrace;
		hipMalloc((void**)&dTrace, sizeof(char)*length*q.size());
		DPwith<<<1,q.size()>>>(dT_seq, dTrace, x - length + 1, length);
		// Direction copy
		char* direction = new char[length*q.size()];
		hipMemcpy(direction, dTrace, sizeof(char)*length*q.size(), hipMemcpyDeviceToHost);	
//		show(direction, db, q, x - length + 1, length);
//		traceback(direction, db, x, y, length);
		delete[] direction;
		hipFree(dT_seq);
		hipFree(dTrace);
	}
}

void SimpleSW::traceback(const char* direction, const Data& db, int x, int y, int length) const{
	// Store the result, get enough size
	char *ans = new char[1024 * 2];
	// Point of result array
	int p = 0;
	int db_point = x;
	// trace point must be most right
	int trace =  length - 1 + y * length;
	// Traceback
	while(trace >= 0){
		switch(direction[trace]){
		case Diagonal:
			ans[p++] = db[db_point--];
			trace -= length + 1;
			break;
		case Vertical:
			ans[p++] = '+';
			trace -= length;
			break;
		case Horizon:
			ans[p++] = '-';
			--trace;
			--db_point;
			break;
		case Zero:	// End
			trace = -1;
		default:
			trace = -1;
		}
	}
	// This array has reverse answer
	for(int i=p-1;i>=0;--i){ cout << ans[i]; }
	printf("  %d ~ %d \n", db_point+1, x);
	delete[] ans;
}

